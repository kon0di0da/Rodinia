#include "hip/hip_runtime.h"
#ifndef _KERNEL_H_
#define _KERNEL_H_

__global__ void
Kernel(Node* g_graph_nodes, int* g_graph_edges, bool* g_graph_mask, bool* g_updating_graph_mask, bool* g_graph_visited, int* g_cost, int no_of_nodes)
{
    int tid = blockIdx.x * MAX_THREADS_PER_BLOCK + threadIdx.x;
    if (tid < no_of_nodes && g_graph_mask[tid])
    {
        g_graph_mask[tid] = false;

        // Use local variables for better performance
        int starting = g_graph_nodes[tid].starting;
        int no_of_edges = g_graph_nodes[tid].no_of_edges;
        int current_cost = g_cost[tid];

        // Use constant memory for g_graph_visited if it doesn't change
        // const bool is_visited = g_graph_visited[tid];

        for (int i = starting; i < (no_of_edges + starting); i++)
        {
            int id = g_graph_edges[i];
            if (!g_graph_visited[id])
            {
                g_cost[id] = current_cost + 1;
                g_updating_graph_mask[id] = true;
            }
        }
    }
}

#endif
